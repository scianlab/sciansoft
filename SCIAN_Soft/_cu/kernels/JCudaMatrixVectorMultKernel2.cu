
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiply(int sizeB, int max, double** A, double* B, double* C, double* Displacement)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < sizeB){
        double sum = 0.0;
        int index_neighbor;
        
        for(int i = 0; i < max; i++) {
            index_neighbor = (int)A[2*tid][i];
            sum = sum + A[2*tid + 1][i]*B[index_neighbor];
        }
        C[tid] = sum;

        Displacement[tid] = B[tid] - C[tid];
    }
}