
#include <hip/hip_runtime.h>
extern "C"
__global__ void surface_area(int size_faces, int* face_verts, double* coord_verts, double* partial_area)
{

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size_faces) {
        int a_index = face_verts[3*tid + 0];
        int b_index = face_verts[3*tid + 1];
        int c_index = face_verts[3*tid + 2];

        double ax = coord_verts[3*a_index + 0];
        double ay = coord_verts[3*a_index + 1];
        double az = coord_verts[3*a_index + 2];

        double bx = coord_verts[3*b_index + 0];
        double by = coord_verts[3*b_index + 1];
        double bz = coord_verts[3*b_index + 2];

        double cx = coord_verts[3*c_index + 0];
        double cy = coord_verts[3*c_index + 1];
        double cz = coord_verts[3*c_index + 2];

        double s1 = ((ax * (by - cy))
                - (bx * (ay - cy))
                + (cx * (ay - by)));

        double s2 = ((ay * (bz - cz))
                - (by * (az - cz))
                + (cy * (az - bz)));

        double s3 = ((az * (bx - cx))
                - (bz * (ax - cx))
                + (cz * (ax - bx)));

        double face_area = 0.5 * sqrt((s1 * s1) + (s2 * s2) + (s3 * s3));

        partial_area[tid] = face_area;
    }
}

extern "C"
__global__ void multiply(int sizeB, int max, double w1, int* A_indexs, double* A_values, double* B, double* C, double* Displacement)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < sizeB){
        double sum = 0.0;
        int index_neighbor;
        
        for(int i = 0; i < max; i++) {
            index_neighbor = A_indexs[max*tid + i];
            sum = sum + A_values[max*tid + i]*B[index_neighbor];
        }
        sum = sum + w1*B[tid];
        C[tid] = sum;

        Displacement[tid] = B[tid] - C[tid];
    }
}