
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiply(int colsA, int sizeB, double** A, double* B, double* C, double* Displacement)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < sizeB){
        double sum = 0.0;
        C[tid] = 0;
        for(int i = 0; i < sizeB; i++) {
            sum = sum + A[tid][i]*B[i];
        }
        C[tid] = sum;

        Displacement[tid] = B[tid] - C[tid];
    }
}